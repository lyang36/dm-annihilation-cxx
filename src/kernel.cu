#include "hip/hip_runtime.h"
#include <cstdio>
#include <rpc/types.h>
#include <rpc/xdr.h>
#include "hip/hip_runtime.h"
#include "math_functions.h"
#include "hip/hip_runtime.h"
#include ""
#include "driver_functions.h"
#include "kernel.h"

namespace kernel_space{
	MAPTYPE * healpixX_GPU;
	MAPTYPE * healpixY_GPU;
	MAPTYPE * healpixZ_GPU;
	
	MAPTYPE * map_GPU;
	
	DMParticle * parts_GPU;

	//the normlized vector
	MAPTYPE * norm_GPU;
	//this is a zero vector, use to zerolize the GPU vec
	MAPTYPE * norm_CPU;
	
	int Npix_;
	int memParts_;
	
}

using namespace kernel_space;

//atomicAdd for double
__device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
                                         (unsigned long long int*)address;
        unsigned long long int old = *address_as_ull, assumed;
            do {
                            assumed = old;
                            old = atomicCAS(address_as_ull, assumed,
                                                                    __double_as_longlong(val +
                                                                                                           __longlong_as_double(assumed)));
                                } while (assumed != old);
                return __longlong_as_double(old);
}

__device__ double SPHKenerl(MAPTYPE d2){
	return exp(-0.5 * d2 / 0.333);
}

__device__ MAPTYPE dotProd(MAPTYPE ax, MAPTYPE ay, MAPTYPE az, MAPTYPE bx, MAPTYPE by, MAPTYPE bz){
	return ax * ax + ay * ay + az * az;
}

//use atomicAdd, may affect perfermance
__global__ void calculateNorm(int Npix, MAPTYPE * healpixX, MAPTYPE * healpixY, MAPTYPE * healpixZ, 
		int numParts, DMParticle * parts, MAPTYPE * norm){
	int pix = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(pix >= Npix){
		return;
	}
	
	int i = 0;
	for(i = 0; i < numParts; i ++){
		if(parts[i].eps < 0){
			continue;
		}
		
		MAPTYPE prod = dotProd(healpixX[pix], healpixY[pix], healpixZ[pix]
		                      , parts[i].velx, parts[i].vely, parts[i].velz);
		
		//could add more constraints here
		if(prod < 0){
			continue;
		}
		
		MAPTYPE d2 = acos(prod) / parts[i].posy;
		if(d2 > 2){
			continue;
		}
		
		d2 = d2 * d2;
		MAPTYPE weight = SPHKenerl(d2);
		atomicAdd(&(norm[i]), weight);
	}
}

//no synchronized, very fast
__global__ void calculateMap(int Npix, MAPTYPE * healpixX, MAPTYPE * healpixY, MAPTYPE * healpixZ, 
		int numParts, DMParticle * parts, MAPTYPE * norm, MAPTYPE * map){
	int pix = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(pix >= Npix){
		return;
	}
	
	int i = 0;
	for(i = 0; i < numParts; i ++){
		if(parts[i].eps < 0){
			continue;
		}
		
		MAPTYPE prod = dotProd(healpixX[pix], healpixY[pix], healpixZ[pix]
		                      , parts[i].velx, parts[i].vely, parts[i].velz);
		
		//could add more constraints here
		if(prod < 0){
			continue;
		}
		
		MAPTYPE d2 = acos(prod) / parts[i].posy;
		if(d2 > 2){
			continue;
		}
		
		d2 = d2 * d2;
		MAPTYPE weight = SPHKenerl(d2);
		map[pix] += weight / norm[i] * parts[i].mass;
	}
}

hipError_t zeroLizeNorm(){
    //zerolize the norm
    hipError_t cudaStatus = hipMemcpy(norm_GPU, norm_CPU, memParts_ * sizeof(MAPTYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying HEALPIX X!\n");
    }
    return cudaStatus;
}

hipError_t initializeCUDA(MAPTYPE * healpixX, MAPTYPE * healpixY, MAPTYPE * healpixZ, int Npix, int memParts){
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }
    Npix_ = Npix;
    memParts_ = memParts;
    

    // Allocate GPU map.
    cudaStatus = hipMalloc((void**)&map_GPU, Npix_ * sizeof(MAPTYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating map memory!");
        return cudaStatus;
    }
    
    // Allocate HEALPIX X.
    cudaStatus = hipMalloc((void**)&healpixX_GPU, Npix_ * sizeof(MAPTYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating healpix x memory!");
        return cudaStatus;
    }
    // Allocate HEALPIX Y.
    cudaStatus = hipMalloc((void**)&healpixY_GPU, Npix_ * sizeof(MAPTYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating healpix y memory!");
        return cudaStatus;
    }
    // Allocate HEALPIX Z.
    cudaStatus = hipMalloc((void**)&healpixZ_GPU, Npix_ * sizeof(MAPTYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating healpix z memory!");
        return cudaStatus;
    }
    
    
    // Allocate GPU Particles.
    cudaStatus = hipMalloc((void**)&parts_GPU, memParts_ * sizeof(DMParticle));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating Particles memory!");
        return cudaStatus;
    }
    
    // Allocate GPU Particle Norm.
    cudaStatus = hipMalloc((void**)&norm_GPU, memParts_ * sizeof(MAPTYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating Particles norm memory!");
        return cudaStatus;
    }
    
    norm_CPU = (MAPTYPE *) calloc(memParts_, sizeof(MAPTYPE));
    zeroLizeNorm();
       
    //copy the HEALPIX X data to GPU
    cudaStatus = hipMemcpy(healpixX_GPU, healpixX, Npix_ * sizeof(MAPTYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying HEALPIX X!\n");
        return cudaStatus;
    }
    
    //copy the HEALPIX Y data to GPU
    cudaStatus = hipMemcpy(healpixY_GPU, healpixY, Npix_ * sizeof(MAPTYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying HEALPIX Y!\n");
        return cudaStatus;
    }
	
    //copy the HEALPIX Z data to GPU
    cudaStatus = hipMemcpy(healpixZ_GPU, healpixZ, Npix_ * sizeof(MAPTYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying HEALPIX Z!\n");
        return cudaStatus;
    }
    return cudaStatus;	
}


void cudaCleaingUp(){
	hipFree(healpixX_GPU);
	hipFree(healpixY_GPU);
	hipFree(healpixZ_GPU);
	hipFree(norm_GPU);
	hipFree(map_GPU);
	hipFree(parts_GPU);
	
	free(norm_CPU);
}

hipError_t calulateMapWithCUDA(MAPTYPE * map, DMParticle * parts, int numParts){
	int blocksize = 512;
	int gridsize = Npix_ / blocksize + 1;
	
	zeroLizeNorm();
    //copy the Map data to GPU
    hipError_t cudaStatus = hipMemcpy(map_GPU, map, Npix_ * sizeof(MAPTYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying HEALPIX X!\n");
        return cudaStatus;
    }
    
    //copy the particle data to GPU
    cudaStatus = hipMemcpy(parts_GPU, parts, memParts_ * sizeof(DMParticle), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying HEALPIX X!\n");
        return cudaStatus;
    }
    
    calculateNorm<<<gridsize, blocksize>>>(Npix_, healpixX_GPU, healpixY_GPU, healpixZ_GPU, 
    		numParts, parts_GPU, norm_GPU);
    cudaStatus = hipDeviceSynchronize();
    if( cudaStatus != hipSuccess){
        fprintf(stderr,"hipDeviceSynchronize error -- sync Norm: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    
    calculateMap<<<gridsize, blocksize>>>(Npix_, healpixX_GPU, healpixY_GPU, healpixZ_GPU, 
    		numParts, parts_GPU, norm_GPU, map_GPU);
    cudaStatus = hipDeviceSynchronize();
    if( cudaStatus != hipSuccess){
        fprintf(stderr,"hipDeviceSynchronize error -- sync map: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    
    //copy map back
     cudaStatus = hipMemcpy(map, map_GPU, Npix_ * sizeof(MAPTYPE), hipMemcpyDeviceToHost);
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMemcpy failed -- copying Map Back!\n");
         return cudaStatus;
     }

     return cudaStatus;
}
