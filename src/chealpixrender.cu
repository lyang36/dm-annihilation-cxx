#include "hip/hip_runtime.h"
#include <cstdio>
//#include <cmath>
#include <rpc/types.h>
#include <rpc/xdr.h>

#include <healpix_base.h>
#include <healpix_map.h>

#include "hip/hip_runtime.h"
#include "math_functions.h"
#include "hip/hip_runtime.h"
#include ""
#include "driver_functions.h"
#include "chealpixrender.h"

#ifndef M_PI
#define M_PI           3.14159265358979323846
#endif

#define PIOVER2  (0.5*M_PI)
#define PI M_PI
#define TWOPI (2.0*M_PI)
#define Z0 (2.0/3.0)
#define TWOTHIRD Z0
#define NS_MAX 8192


float * d_map;
renderpart * d_parts;
int nside_;

hipError_t initializeCUDA(int nside, int numofparts){

    int count = 0;
	int i = 0;
    hipError_t cudaStatus;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return hipErrorNotReady;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return hipErrorNotReady;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");

    nside_ = nside;
    int npix = 12 * nside * nside;
    cudaStatus = hipMalloc((void**)&d_map, npix * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating HEALPix map memory!\n");
        return cudaStatus;
    }
    
    //clear the memory
    hipMemset(d_map, 0, npix * sizeof(float));
    
    cudaStatus = hipMalloc((void**)&d_parts, numofparts * sizeof(renderpart));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating Render Particles memory!\n");
        return cudaStatus;
    }
    return cudaStatus;
}


hipError_t calculateMapByGPU(renderpart * parts, int num_of_parts){
    int blocks = num_of_parts;
    if(num_of_parts == 0){
        return hipSuccess;
    }
    //cuda mem copy
    //copy particle data to GPU
    hipError_t cudaStatus = hipMemcpy(d_parts, parts, num_of_parts * sizeof(renderpart), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying particle data to device!\n");
        return cudaStatus;
    }
    
    calcfluxGPU<<<blocks, NUM_THREADS_PER_BLOCK>>>(nside_,
                                                   d_map,
                                                   num_of_parts,
                                                   d_parts);
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("hipDeviceSynchronize error: %s\n", hipGetErrorString(cudaStatus));
    }
    return cudaStatus;
}

hipError_t getCUDAMap(float * map){
    int npix = 12 * nside_ * nside_;
    hipError_t cudaStatus = hipMemcpy(map, d_map, npix * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying map data to host!\n");

    }
    return cudaStatus;
}

void cudaCleaingUp(){
    hipFree(d_map);
	hipFree(d_parts);
}

__device__ float SPHKenerl(float d2){
	return exp(-0.5 * d2 / 0.333);
}

//(x1, y1, z1), (x2, y2, z2) must be normalized
__device__ float flux(healpix_par &par, float x1, float y1, float z1, float x2, float y2, float z2, float r_angle){
    float prod = x1 * x2 + y1 * y2 + z1 * z2;
    if(prod > 1) prod = 1;
    float atheta = acos(prod);
    if(atheta > (par.theta_per_pix / 2.0 + r_angle)){
        return 0;
    }
    float d2 =  atheta / r_angle;
    d2 *= d2;
    return SPHKenerl(d2);
}


// 27 flops
__device__ void angle2pix(healpix_par &par,
                          float z,
                          float phi,
                          int & iring,
                          int & icol,
                          int & ipix){
    
    int jp, jm;
    float  tt, tp, tmp;
    int ir, ip, kshift;
    
    float za = fabs(z);
    if( phi >= TWOPI)  phi = phi - TWOPI;
    if (phi < 0.)     phi = phi + TWOPI;
    tt = phi / PIOVER2;//  ! in [0,4)
    
    if( za <= Z0 ) {
        jp = (int)floor(par.nside*(0.5 + tt - z*0.75)); /*index of ascending edge line*/
        jm = (int)floor(par.nside*(0.5 + tt + z*0.75)); /*index of descending edge line*/
        ir = par.nside + 1 + jp - jm;// ! in {1,2n+1} (ring number counted from z=2/3)
        iring = ir + par.nside; // ! in {1,3n+1} (ring number counted from z=1)
        
        kshift = 0;
        if (ir % 2==0.) kshift = 1;// ! kshift=1 if ir even, 0 otherwise
        ip = ( ( jp+jm - par.nside + kshift + 1 ) / 2 );// ! in {0,4n)
        if( ip>par.nl4 ) ip = ip - par.nl4;
        icol = ip;
        ipix = par.ncap + par.nl4*(ir-1) + ip ;
    }else {
        tp = tt - floor(tt);//      !MOD(tt,1.d0)
        tmp = sqrt( 3.*(1. - za) );
        
        jp = (int)floor( par.nside * tp * tmp );// ! increasing edge line index
        jm = (int)floor( par.nside * (1. - tp) * tmp );// ! decreasing edge line index
        
        ir = jp + jm + 1;//        ! ring number counted from the closest pole
        ip = (int)floor( tt * ir );// ! in {0,4*ir)
        
        if( ip>4*ir ) ip = ip - 4*ir;
        
        ipix = 2*ir*(ir-1) + ip;
        iring = ir;
        icol = ip;
        if( z<=0. ) {
            ipix = par.npix - 2*ir*(ir+1) + ip;
            iring = par.nl4 - ir;
        }
    }
}

//6 flops
__device__ void pix2vec(healpix_par &par, int r, int c, float &x, float &y, float &z, float &ct, float &phi){
    float sintheta;
    if(r <= par.nside){
        ct  = 1.0 - 4.0 * r * r / (float) par.npix;
        phi = (c + 0.5) / (2.0 * r) * M_PI;
    }else if (r < par.nl3){
        ct  = 2.0 / 3.0 * (2.0 * par.nside - r) / (float) par.nside;
        phi = (c + 0.5 * (1 - (r + par.nside) % 2)) / (2.0 * par.nside) * M_PI;
    }else{
        float cr = par.nl4 - r;
        ct = -1.0 + 4.0 * cr * cr / (float)par.npix;
        phi = (c + 0.5) / (2 * cr) * M_PI;
    }
    sintheta = sqrt(1 - ct * ct);
    x = sintheta * cos(phi);
    y = sintheta * sin(phi);
    z = ct;
}



//4 flops
__device__ int cr2pix(healpix_par &par, int col, int ring){
    if(ring <= par.nside){
        return 2 * ring * (ring - 1) + col;
    }else if(ring  < par.nl3){
        return par.nl4 * ring - par.nl2 * par.nside - par.nl2 + col;
    }else{
        int r = par.nl4 - ring;
        return par.npix - (2 * r * (r + 1)) + col;
    }
}

//4 flops
__device__ int pix2icol(healpix_par &par, int ring, int pix){
    return pix - cr2pix(par, 0, ring);
}

//3 flops
__device__ int pix2ring(healpix_par &par, int ipix){
    if(ipix < par.ncap + par.nl4){
        return ((sqrt(2.0 * ipix + 1) + 1) / 2);
    }else if (ipix < 10 * par.nsidesq - par.nl4){
        return ((ipix - 2 * par.nsidesq - par.nl2)/par.nl4) + par.nside + 1;
    }else{
        //floor((sqrt(2*(12 * ns * ns - pix - 1) + 1) + 1) / 2)
        return par.nl4 - floor((sqrt(2.0*(par.npix - 1- ipix) + 1) + 1) / 2);
    }
}

//4 flops
//get the ring num of certain z
__device__ int ring_above (long nside_, float z){
    float az=abs(z);
    if (az>TWOTHIRD) // polar caps
    {
        int iring = int(nside_*sqrt(3*(1-az)));
        return (z>0) ? iring : 4*nside_-iring-1;
    }
    else // ----- equatorial region ---------
        return int(nside_*(2-1.5*z));
}

//count flops
__global__ void calcfluxGPU(int nside,
                       float * map,
                       int numOfParts,
                       renderpart * parts){
    healpix_par pars;
    __shared__ float pixval[NUM_THREADS_PER_BLOCK];
    
    int icol, iring, dc, rmin, rmax, dr, ipix;
    bool isNorthPolarIn = false;
    bool isSouthPolarIn = false;
    int numPix, npixNorthPole = 0, npixSouthPole = 0, totalPix;
    int numPixPerThread;
    int startPix = 0;
    float z;
    float x, y;
    float  phi;
    float norm = 0;
    
    if(blockIdx.x >= numOfParts){
        return;
    }
    
    renderpart particle = parts[blockIdx.x];

    
    //6 flops
    pars.nl2 = 2*nside;
    pars.nl4 = 4*nside;
    pars.nl3 = 3*nside;
    pars.nside = nside;
    pars.nsidesq = nside * nside;
    pars.ncap  = pars.nl2*(nside-1);
            // ! number of pixels in the north polar cap
    pars.npix  = 12*nside*nside;
    pars.theta_per_pix = 2 * M_PI / pars.nl4;
    x = particle.x;
    y = particle.y;
    z = particle.z;
    //theta = particle.theta;
    phi = particle.phi;
    float za = fabs(z);
    
    //10 flops
    float rlat1 = particle.theta - particle.angular_radius;
    float zmax = cos(rlat1);
    rmin = ring_above (nside, zmax) + 1;
    float rlat2 = particle.theta + particle.angular_radius;
    float zmin = cos(rlat2);
    rmax = ring_above (nside, zmin) + 1;
    
    //27 flops
    //get the phi and col of theta and phi
    angle2pix(pars, z, particle.phi, iring, icol, ipix);
    
    dc = particle.angular_radius / pars.theta_per_pix + 1;
    dr = rmax - rmin;
    
    numPix = (2 * dc + 1) * dr;
    isNorthPolarIn = (rlat1 <= 0.0);
    isSouthPolarIn = (rlat2 >= M_PI);
    
    if(isNorthPolarIn)
            npixNorthPole = 2 * rmin * (rmin - 1);
    
    if(isSouthPolarIn)
            npixSouthPole = pars.npix - 2 * (pars.nl4 - rmax) * (pars.nl4 - rmax - 1);

    totalPix = numPix + npixNorthPole + npixSouthPole;
    
    numPixPerThread = totalPix / NUM_THREADS_PER_BLOCK
        + (totalPix % NUM_THREADS_PER_BLOCK == 0) ? 0 : 1;
    startPix = 0;
    
    bool isIgnored = false;

    int c0;
    int p = 0;
    int pr = 0;
    int pc = 0;
    float weight = 0;
    int k = 0;
    for(int i = 0; i < numPixPerThread; i++){
        pixval[threadIdx.x] = 0;
        __syncthreads();
        
        k = threadIdx.x + startPix;
        if( k > totalPix){
            break;
        }
        
        //calculate the pixel id
        if (k < npixNorthPole){
            p=k;
            pr = pix2ring(pars, p);
            pc = pix2icol(pars, pr, p);
        }
        else if (k < npixNorthPole + npixSouthPole){
            p = pars.npix - (k - npixNorthPole) - 1;
            pr = pix2ring(pars, p);
            pc = pix2icol(pars, pr, p);
        }else{
            int np = k - npixNorthPole - npixSouthPole;
            pr = np / (2 * dc +1)+rmin;
            if(pr < 1 || pr > pars.nl4){
                isIgnored = true;
            }else{
                if(pr <= pars.nside){
                    c0 = (int)(2 * (phi - particle.angular_radius) * pr / M_PI) - 1;
                }
                else if(pr < pars.nl3){
                    c0 = int(phi - particle.angular_radius)
                        / pars.theta_per_pix - 1;
                }else{
                    c0 = (int)(2 * (phi - particle.angular_radius) *
                        (pars.nl4 - pr) / M_PI) - 1;
                }
                pc = np % (2 * dc +1)+c0;
                if((pc < 0) ||
                   (pr <= pars.nside && pc >= 4 * pr) ||
                   (pr < pars.nl3 && pc >= pars.nl4) ||
                   (pr >= pars.nl3 && pc > 4 * (pars.nl4 - pr))){
                    isIgnored = true;
                }else{
                    p = cr2pix(pars, pc, pr);
                }
            }

        }
 
        //calculate the value
        if(!isIgnored){
            float x1, y1, z1, ct, phi;
            pix2vec(pars, pr, pc, x1, y1, z1, ct, phi);
            weight = pixval[i] = flux(pars, x1, y1, z1,
                            x, y, z,
                            particle.angular_radius);
        }
        
        
        //calculate the norm (reduce-sweeping algorithm)
        //up-sweeping
        int halfThreadNum = NUM_THREADS_PER_BLOCK / 2;
        while(halfThreadNum > 0){
            if(threadIdx.x < halfThreadNum){
                pixval[threadIdx.x] += pixval[halfThreadNum + threadIdx.x];
            }
            __syncthreads();
            halfThreadNum /= 2;
        }
        //down-sweeping
        halfThreadNum = 1;
        while(halfThreadNum < NUM_THREADS_PER_BLOCK){
            if(threadIdx.x < halfThreadNum){
                pixval[halfThreadNum + threadIdx.x] = pixval[threadIdx.x];
            }
            __syncthreads();
            halfThreadNum *= 2;
        }
        norm = pixval[threadIdx.x];
        
        //calculated the result and record them to the global memory
        if(!isIgnored){
            if(numPixPerThread == 1){
                //map[p]  = weight / norm;
                atomicAdd(map + p, weight * particle.flux / norm);
            }
        }
        
        //continue to next block
        startPix += NUM_THREADS_PER_BLOCK;
    }
    
    if(numPixPerThread == 1){
        return;
    }
    
    startPix = 0;
    for(int i = 0; i < numPixPerThread; i++){
        //calculate the pixel id
        if (k < npixNorthPole){
            p=k;
            pr = pix2ring(pars, p);
        }
        else if (k < npixNorthPole + npixSouthPole){
            p = pars.npix - (k - npixNorthPole) - 1;
            pr = pix2ring(pars, p);
            
        }else{
            int np = k - npixNorthPole - npixSouthPole;
            pr = np / (2 * dc +1)+rmin;
            if(pr < 1 || pr > pars.nl4){
                isIgnored = true;
            }else{
                if(pr <= pars.nside){
                    c0 = (int)(2 * (phi - particle.angular_radius) * pr / M_PI) - 1;
                }
                else if(pr < pars.nl3){
                    c0 = int(phi - particle.angular_radius)
                    / pars.theta_per_pix - 1;
                }else{
                    c0 = (int)(2 * (phi - particle.angular_radius) *
                               (pars.nl4 - pr) / M_PI) - 1;
                }
                pc = np % (2 * dc +1)+c0;
                if((pc < 0) ||
                   (pr <= pars.nside && pc >= 4 * pr) ||
                   (pr < pars.nl3 && pc >= pars.nl4) ||
                   (pr >= pars.nl3 && pc > 4 * (pars.nl4 - pr))){
                    isIgnored = true;
                }else{
                    p = cr2pix(pars, pc, pr);
                }
            }
            
        }
        
        //calculate the value
        if(!isIgnored){
            float x1, y1, z1, ct, phi;
            pix2vec(pars, pr, pc, x1, y1, z1, ct, phi);
            weight = flux(pars, x1, y1, z1,
                          x, y, z,
                          particle.angular_radius);
            atomicAdd(map + p, weight * particle.flux / norm);
        }

    }
}
