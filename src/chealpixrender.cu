#include "hip/hip_runtime.h"
#include <cstdio>
//#include <cmath>
#include <rpc/types.h>
#include <rpc/xdr.h>

#include <healpix_base.h>
#include <healpix_map.h>

#include "hip/hip_runtime.h"
#include "math_functions.h"
#include "hip/hip_runtime.h"
#include ""
#include "driver_functions.h"
#include "chealpixrender.h"

#ifndef M_PI
#define M_PI           3.14159265358979323846
#endif

#define PIOVER2  (0.5*M_PI)
#define PI M_PI
#define TWOPI (2.0*M_PI)
#define Z0 (2.0/3.0)
#define TWOTHIRD Z0
#define NS_MAX 8192


MAPTYPE * d_map;
renderpart * d_parts;
healpix_par params;
int nside_;

hipError_t initializeCUDA(int nside, int numofparts){

    int count = 0;
	int i = 0;
    hipError_t cudaStatus;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return hipErrorNotReady;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return hipErrorNotReady;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");

    nside_ = nside;
    params.setup(nside);
    
    int npix = 12 * nside * nside;
    cudaStatus = hipMalloc((void**)&d_map, npix * sizeof(MAPTYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating HEALPix map memory!\n");
        return cudaStatus;
    }
    
    //clear the memory
    hipMemset(d_map, 0, npix * sizeof(MAPTYPE));
    
    cudaStatus = hipMalloc((void**)&d_parts, numofparts * sizeof(renderpart));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating Render Particles memory!\n");
        return cudaStatus;
    }
    return cudaStatus;
}


hipError_t calculateMapByGPU(renderpart * parts, int num_of_parts){
    int blocks = num_of_parts;
    if(num_of_parts == 0){
        return hipSuccess;
    }
    //cuda mem copy
    //copy particle data to GPU
    hipError_t cudaStatus = hipMemcpy(d_parts, parts, num_of_parts * sizeof(renderpart), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying particle data to device!\n");
        return cudaStatus;
    }
    
    calcfluxGPU<<<blocks, NUM_THREADS_PER_BLOCK>>>(params,
                                                   d_map,
                                                   num_of_parts,
                                                   d_parts);
    

    return cudaStatus;
}

hipError_t getCUDAMap(MAPTYPE * map){

	hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("hipDeviceSynchronize error: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
    }

    int npix = 12 * nside_ * nside_;
    cudaStatus = hipMemcpy(map, d_map, npix * sizeof(MAPTYPE), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying map data to host!\n");

    }
    return cudaStatus;
}

void cudaCleaingUp(){
    hipFree(d_map);
	hipFree(d_parts);
}

void renderpart::setup(int nside){
    healpix_par params_;
    params_.setup(nside);
    
    //the ring number
    int iring;
    int icol;
   
    if( phi >= TWOPI)  phi = phi - TWOPI;
    if (phi < 0.)     phi = phi + TWOPI;    
    


    int ipix, dr;
    MAPTYPE rlat1 = theta - 2.0 * angular_radius;
    MAPTYPE zmax = cos(rlat1);
    rmin = ring_above (nside, zmax) - 1;
    MAPTYPE rlat2 = theta + 2.0 * angular_radius;
    MAPTYPE zmin = cos(rlat2);
    rmax = ring_above (nside, zmin) + 1;
    angle2pix(params_, z, phi, iring, icol, ipix);
    
    dc = (int)(4.0 * angular_radius / params.theta_per_pix +0.5) + 2;
    dr = rmax - rmin;
    
    numPix = (2 * dc + 1) * dr;
    
    bool isNorthPolarIn = (rlat1 <= 0.0);
    bool isSouthPolarIn = (rlat2 >= M_PI);
    
    npixNorthPole = 0;
    if(isNorthPolarIn)
        npixNorthPole = 2 *
            rmin * (rmin - 1);
    
    npixSouthPole = 0;
    if(isSouthPolarIn)
        npixSouthPole = params_.npix -
            2 * (params_.nl4 - rmax) *
            (params_.nl4 - rmax - 1);
    
    
}


__device__ MAPTYPE SPHKenerl(MAPTYPE d2){
    //test
	return exp(-0.5 * d2 / 0.333);
    //return exp(- 10 * d2 );

}

//(x1, y1, z1), (x2, y2, z2) must be normalized
__device__ MAPTYPE flux(healpix_par &par, MAPTYPE x1, MAPTYPE y1, MAPTYPE z1, MAPTYPE x2, MAPTYPE y2, MAPTYPE z2, MAPTYPE r_angle){
    MAPTYPE prod = x1 * x2 + y1 * y2 + z1 * z2;
    //test
    //return 1.0;

    if(prod > 1) prod = 1;
    if(prod < -1) prod = -1;
    MAPTYPE atheta = acos(prod);
    if(atheta > (par.theta_per_pix / 2.0 + r_angle * 2)){
        return 0;
    }

    //test
    //return 1.0;

    MAPTYPE d2 =  atheta / r_angle;
    d2 *= d2;
    return SPHKenerl(d2);
}


// 27 flops
__host__ __device__ void angle2pix(healpix_par &par,
                          MAPTYPE z,
                          MAPTYPE phi,
                          int & iring,
                          int & icol,
                          int & ipix){
    
    int jp, jm;
    MAPTYPE  tt, tp, tmp;
    int ir, ip, kshift;
    
    MAPTYPE za = fabs(z);
    if( phi >= TWOPI)  phi = phi - TWOPI;
    if (phi < 0.)     phi = phi + TWOPI;
    tt = phi / PIOVER2;//  ! in [0,4)
    
    if( za <= Z0 ) {
        jp = (int)floor(par.nside*(0.5 + tt - z*0.75)); /*index of ascending edge line*/
        jm = (int)floor(par.nside*(0.5 + tt + z*0.75)); /*index of descending edge line*/
        ir = par.nside + 1 + jp - jm;// ! in {1,2n+1} (ring number counted from z=2/3)
        iring = ir + par.nside; // ! in {1,3n+1} (ring number counted from z=1)
        
        kshift = 0;
        if (ir % 2==0.) kshift = 1;// ! kshift=1 if ir even, 0 otherwise
        ip = ( ( jp+jm - par.nside + kshift + 1 ) / 2 );// ! in {0,4n)
        if( ip>par.nl4 ) ip = ip - par.nl4;
        icol = ip;
        ipix = par.ncap + par.nl4*(ir-1) + ip ;
    }else {
        tp = tt - floor(tt);//      !MOD(tt,1.d0)
        tmp = sqrt( 3.*(1. - za) );
        
        jp = (int)floor( par.nside * tp * tmp );// ! increasing edge line index
        jm = (int)floor( par.nside * (1. - tp) * tmp );// ! decreasing edge line index
        
        ir = jp + jm + 1;//        ! ring number counted from the closest pole
        ip = (int)floor( tt * ir );// ! in {0,4*ir)
        
        if( ip>4*ir ) ip = ip - 4*ir;
        
        ipix = 2*ir*(ir-1) + ip;
        iring = ir;
        icol = ip;
        if( z<=0. ) {
            ipix = par.npix - 2*ir*(ir+1) + ip;
            iring = par.nl4 - ir;
        }
    }
}

//6 flops
__host__ __device__ void pix2vec(healpix_par &par, int r, int c, MAPTYPE &x, MAPTYPE &y, MAPTYPE &z, MAPTYPE &ct, MAPTYPE &phi){
    MAPTYPE sintheta;
    if(r <= par.nside){
        ct  = 1.0 - 4.0 * r * r / (MAPTYPE) par.npix;
        phi = (c + 0.5) / (2.0 * r) * M_PI;
    }else if (r < par.nl3){
        ct  = 2.0 / 3.0 * (2.0 * par.nside - r) / (MAPTYPE) par.nside;
        phi = (c + 0.5 * (1 - (r + par.nside) % 2)) / (2.0 * par.nside) * M_PI;
    }else{
        MAPTYPE cr = par.nl4 - r;
        ct = -1.0 + 4.0 * cr * cr / (MAPTYPE)par.npix;
        phi = (c + 0.5) / (2 * cr) * M_PI;
    }
    sintheta = sqrt(1 - ct * ct);
    x = sintheta * cos(phi);
    y = sintheta * sin(phi);
    z = ct;
}



//4 flops
__host__ __device__ int cr2pix(healpix_par &par, int col, int ring){
    if(ring <= par.nside){
        return 2 * ring * (ring - 1) + col;
    }else if(ring  < par.nl3){
        return par.nl4 * ring - par.nl2 * par.nside - par.nl2 + col;
    }else{
        int r = par.nl4 - ring;
        return par.npix - (2 * r * (r + 1)) + col;
    }
}

//4 flops
__host__ __device__ int pix2icol(healpix_par &par, int ring, int pix){
    return pix - cr2pix(par, 0, ring);
}

//3 flops
__host__ __device__ int pix2ring(healpix_par &par, int ipix){
    if(ipix < par.ncap + par.nl4){
        return (int)((sqrt(2.0 * ipix + 1.0) + 1.0) / 2.0);
    }else if (ipix < 10 * par.nsidesq - par.nl4){
        return ((ipix - 2 * par.nsidesq - par.nl2)/par.nl4) + par.nside + 1;
    }else{
        //floor((sqrt(2*(12 * ns * ns - pix - 1) + 1) + 1) / 2)
        return par.nl4 - (int) floor((sqrt(2.0*(par.npix - 1.0 - ipix) + 1.0) + 1.0) / 2.0);
    }
}

//4 flops
//get the ring num of certain z
__host__ __device__ int ring_above (long nside_, MAPTYPE z){
    MAPTYPE az=abs(z);
    if (az>TWOTHIRD) // polar caps
    {
        int iring = (int)(nside_*sqrt(3*(1-az)));
        return (z>0) ? iring : (4*nside_-iring-1);
    }
    else // ----- equatorial region ---------
        return (int)(nside_*(2-1.5f*z));
}

__device__ void getPixIdThread(
                healpix_par &params, renderpart &particle,
                int k, int dc, int rmin, MAPTYPE phi,
                int  npixNorthPole, int npixSouthPole,
                int &pr, int &pc, int &p){
        int c0 = 0;
        p = -1;
        if (k < (npixNorthPole)){
                
                p=k;
                pr = pix2ring(params, p);
                pc = pix2icol(params, pr, p);
                
        }
        else if (k < (npixNorthPole + npixSouthPole)){
                
                p = params.npix - (k - npixNorthPole) - 1;
                pr = pix2ring(params, p);
                pc = pix2icol(params, pr, p);
                
        }else{
                int np = k - npixNorthPole - npixSouthPole;
                pr = np / (2 * dc +1)+rmin;
                if(pr < 1 || pr > params.nl4){
                        p = -1;
                }else{
                    int  npixatthisring = params.nl4;
                        //c0 = (int)((phi - 2.0 * particle.angular_radius)
                        //             / params.theta_per_pix - 1);
                    if(pr <= params.nside){
                        //c0 = (int)floor(2 * (phi - (2.0 * particle.angular_radius)) * pr / M_PI
                        //                - 0.5)
                        //                + 1;
                        npixatthisring = 4 * pr;
                    }
                    else if(pr < params.nl3){
                        //c0 = (int)floor((phi - 2.0 * particle.angular_radius)
                        //           / params.theta_per_pix - 0.5) + 1;
                        npixatthisring = params.nl4;
                    }else{
                        //c0 = (int)floor(2 * (phi - 2.0 * particle.angular_radius) *
                        //           (params.nl4 - pr) / M_PI 
                        //           - 0.5) + 1;
                        npixatthisring = 4 * (params.nl4 - pr);
                    }
				
					c0 = floor(npixatthisring/TWOPI*(phi) - 0.5) - dc;
                    //int ip_hi = floor(npixatthisring/TWOPI*(phi0+dphi) - 0.5);


                    pc = np % (2 * dc +1)+c0;
                    if(pc < 0) pc += npixatthisring;
                    if(pc > npixatthisring) pc = pc % npixatthisring;
                    p = cr2pix(params, pc, pr);
                    
              }
       }
       if((p < 0) || (p > params.npix)){
              p = -1;
       }
                
}


__device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
                                         (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
       assumed = old;
       old = atomicCAS(address_as_ull, assumed,
       __double_as_longlong(val +__longlong_as_double(assumed)));
       } while (assumed != old);
    return __longlong_as_double(old);
}



//count flops
__global__ void calcfluxGPU(
                            healpix_par params,
                            MAPTYPE * map,
                            int numOfParts,
                            renderpart * parts
                            ){

    __shared__ MAPTYPE pixval[NUM_THREADS_PER_BLOCK];
    __shared__ renderpart listOfParticles[NUM_THREADS_PER_BLOCK];
    renderpart particle;
    
    if(blockIdx.x >= numOfParts){
        return;
    }
    
    ////////////////////////Read Particles/////////////////////////
    //listOfParticles[blockIdx.x] = particle;
    if(threadIdx.x == 0){
        particle = parts[blockIdx.x];
        listOfParticles[0] = particle;
    }
    __syncthreads();
    
    //down-sweeping
    int halfThreadNum = 1;
    while(halfThreadNum < NUM_THREADS_PER_BLOCK){
        if(threadIdx.x < halfThreadNum){
            listOfParticles[halfThreadNum + threadIdx.x]
                = listOfParticles[threadIdx.x];
        }
        __syncthreads();
        halfThreadNum *= 2;
    }
    particle = listOfParticles[threadIdx.x];
    ////////////////////////////////////////////////////////////////
    
    int rmin;
    int dc;
    int numPix, npixNorthPole, npixSouthPole, totalPix;
    
    int numPixPerThread;
    int startPix = 0;
    
    MAPTYPE x, y, z;
    
    MAPTYPE  phi;
    MAPTYPE norm = 0;

    
    dc = particle.dc;
    x = particle.x;
    y = particle.y;
    z = particle.z;
    phi = particle.phi;
    numPix = particle.numPix;
    npixNorthPole = particle.npixNorthPole;
    npixSouthPole = particle.npixSouthPole;
    rmin = particle.rmin;
    
    MAPTYPE za = fabs(z);
    
    totalPix = numPix + npixNorthPole + npixSouthPole;
    
    
    numPixPerThread = totalPix / NUM_THREADS_PER_BLOCK
        + ((totalPix % NUM_THREADS_PER_BLOCK == 0) ? 0 : 1);
    
    startPix = 0;
    

    //int c0;
    int p = 0;
    int pr = 0;
    int pc = 0;
    MAPTYPE weight = 0;
    int k = 0; 
    norm = 0.0;
    
    for(int i = 0; i < numPixPerThread; i++){
        weight = 0;

        k = threadIdx.x + startPix;
        if( k < totalPix){
        
            //calculate the pixel id
                                
            getPixIdThread(params, particle,
                    k, dc, rmin, phi,
                    npixNorthPole, npixSouthPole,
                    pr, pc, p);
                
            if(p != -1){ 
            
                //calculate the value
                MAPTYPE x1, y1, z1, ct, phi1;
                pix2vec(params, pr, pc, x1, y1, z1, ct, phi1);
                //test
                weight = flux(params, x1, y1, z1,
                          x, y, z,
                          particle.angular_radius);
                //pixval[threadIdx.x] = weight;
            }
        }
        pixval[threadIdx.x] = weight;
        
        /////////////////////Calculating Norm////////////////////////
        //calculate the norm (reduce-sweeping algorithm)
        //up-sweeping
        __syncthreads();
        halfThreadNum = NUM_THREADS_PER_BLOCK / 2;
        while(halfThreadNum > 0){
            if(threadIdx.x < halfThreadNum){
                pixval[threadIdx.x] += pixval[halfThreadNum + threadIdx.x];
            }
            __syncthreads();
            halfThreadNum /= 2;
        }
        //down-sweeping
        halfThreadNum = 1;
        while(halfThreadNum < NUM_THREADS_PER_BLOCK){
            if(threadIdx.x < halfThreadNum){
                pixval[halfThreadNum + threadIdx.x] = pixval[threadIdx.x];
            }
            __syncthreads();
            halfThreadNum *= 2;
        }
        norm += pixval[threadIdx.x];
        ////////////////////////////////////////////////////////////
        
        //calculated the result and record them to the global memory
        
        if(numPixPerThread == 1){
			if((norm > 0) && (p != -1)){
				//test
                MAPTYPE fofp = weight * particle.flux / norm;
                atomicAdd(map + p, fofp);
				//atomicAdd(map + p, weight / norm);
				//map[p] = particle.flux;
			}
        }

        
        //continue to next block
        startPix += NUM_THREADS_PER_BLOCK;
    }
    
    if(numPixPerThread <= 1){
        return;
    }
    
    startPix = 0;
    for(int i = 0; i < numPixPerThread; i++){
        k = threadIdx.x + startPix;
        if( k >= totalPix){
        }else{
            
            //calculate the pixel id
            getPixIdThread(params, particle,
                    k, dc, rmin, phi,
                    npixNorthPole, npixSouthPole,
                    pr, pc, p);
            
            if(p != -1 ){
                //calculate the value
                MAPTYPE x1, y1, z1, ct, phi1;
                pix2vec(params, pr, pc, x1, y1, z1, ct, phi1);
            
			    if(norm > 0){
                    //test 
                    weight = flux(params, x1, y1, z1,
                          x, y, z,
                          particle.angular_radius);

				    //test
            	    atomicAdd(map + p, weight * particle.flux / norm);
				    //atomicAdd(map + p, weight / norm);
			    }

            }
        }
        
        startPix += NUM_THREADS_PER_BLOCK;

    }
}
